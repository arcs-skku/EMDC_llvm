#include "hip/hip_runtime.h"
//====================================================================================================100
//		UPDATE
//====================================================================================================100

//    2006.03   Rob Janiczek
//        --creation of prototype version
//    2006.03   Drew Gilliam
//        --rewriting of prototype version into current version
//        --got rid of multiple function calls, all code in a  
//         single function (for speed)
//        --code cleanup & commenting
//        --code optimization efforts   
//    2006.04   Drew Gilliam
//        --added diffusion coefficent saturation on [0,1]
//		2009.12 Lukasz G. Szafaryn
//		-- reading from image, command line inputs
//		2010.01 Lukasz G. Szafaryn
//		--comments

//====================================================================================================100
//	DEFINE / INCLUDE
//====================================================================================================100

#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <algorithm>
#include <vector>
#include <string>
#include <bitset>
#include <bemps.hpp>
#include <chrono>
#include <iostream>

#include "define.c"
#include "extract_kernel.cu"
#include "prepare_kernel.cu"
#include "reduce_kernel.cu"
#include "srad_kernel.cu"
#include "srad2_kernel.cu"
#include "compress_kernel.cu"
#include "graphics.c"
#include "resize.c"
#include "timer.c"

#include "cufile.h"
#include <unistd.h>
#include <error.h>
#include <fcntl.h>
#include <errno.h>

#include "device.c"				// (in library path specified to compiler)	needed by for device functions

using namespace std::chrono;

#define page_size 4096
#define VA_block 2097152

#define CUDA_CHECK(val) { \
	if (val != hipSuccess) { \
		fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(val), __LINE__, __FILE__); \
		exit(val); \
	} \
}

size_t page_align (size_t mem){
	if((mem % 2097152) != 0){
		return (2097152 * (mem / 2097152 + 1));
	}
	else{
		return mem;
	}
}

bool full; // 1 = fully secured
int priority;

struct Parameters{
	void* devPtr;
	size_t count;
	hipMemoryAdvise advice;
	int device;
	size_t alloc_size; // 디바이스 메모리에 올릴 페이지 크기
	// std::bitset<kernel_num> bit; // liveness check
};

std::vector<Parameters> mem_list;

void init (fp* arr, int size){
	for(int i = 0; i < size / sizeof(fp); i++){
		arr[i] = 0;
	}
}

void task_monitoring(hipEvent_t event, int tid, long orig_alloc_mem, size_t membytes){
	long update_mem = 0;
	long tmp_mem = 0;
	update_mem = bemps_extra_task_mem(tid);
	tmp_mem = update_mem;
	if(full != 1){
		while(1){
			bool chk_former_task = 0;
			update_mem = bemps_extra_task_mem(tid);
			// hipStream_t s_e;
			// CUDA_CHECK(hipStreamCreate(&s_e));
			if(orig_alloc_mem != update_mem){
				chk_former_task = 1;
			}
			if(hipEventQuery(event) == hipSuccess){
				printf("Kernel End\n");
				break;
			}
			if((chk_former_task == 1) && (full != 1)){
				if(update_mem == membytes){
					full = 1;
				}
				if(full == 1){
					priority = -5;
					hipStream_t s_e;
					CUDA_CHECK(hipStreamCreateWithPriority(&s_e, 0, priority));
					printf("Hello\n");
					int loop = 0;
					for(Parameters ret : mem_list){
						CUDA_CHECK(hipMemAdvise(ret.devPtr, ret.alloc_size, ret.advice, ret.device));
						if(loop < 5){
							CUDA_CHECK(hipMemPrefetchAsync(ret.devPtr, ret.alloc_size, ret.device, s_e));
						}
						loop++;
					}
					CUDA_CHECK(hipStreamSynchronize(s_e));
					CUDA_CHECK(hipStreamDestroy(s_e));
					break;
				}
			}
			// CUDA_CHECK(hipStreamDestroy(s_e));
		}
	}
}

//====================================================================================================100
//	MAIN FUNCTION
//====================================================================================================100

int main(int argc, char *argv []){

	//================================================================================80
	// 	VARIABLES
	//================================================================================80
	int tid = atoi(argv[5]);
	
	struct timespec specific_time;
	struct tm *now;
	int millsec;
	clock_gettime( CLOCK_REALTIME, &specific_time);
	now = localtime(&specific_time.tv_sec);
	millsec = specific_time.tv_nsec;
  
	millsec = floor (specific_time.tv_nsec/1.0e6);
  
	printf("TID: %d Application begin, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
		now->tm_mon + 1, now->tm_mday, now->tm_hour, 
		now->tm_min, now->tm_sec, millsec);

	size_t ef_mem;
	int ef_cnt = 0;

	int ret_dev_id;

	Parameters ret1;
	Parameters ret2;
	Parameters ret3;
	Parameters ret4;
	Parameters ret5;
	Parameters ret6;
	Parameters ret7;
	Parameters ret8;
	Parameters ret9;
	Parameters ret10;
	Parameters ret11;
	Parameters ret12;
	Parameters ret13;
	
	// time
	long long time0;
	long long time1;
	long long time2;
	long long time3;
	long long time4;
	long long time5;
	long long time6;
	long long time7;
	long long time8;
	long long time9;
	long long time10;
	long long time11;
	long long time12;

	time0 = get_time();

	fp* tmp_image_ori, *tmp_image;

    // inputs image, input paramenters
    fp* image_ori;																// originalinput image
	int image_ori_rows;
	int image_ori_cols;
	long image_ori_elem;

    // inputs image, input paramenters
    fp* image;															// input image
    int Nr,Nc;													// IMAGE nbr of rows/cols/elements
	long Ne;

	// algorithm parameters
    int niter;																// nbr of iterations
    fp lambda;															// update step size

    // size of IMAGE
	int r1,r2,c1,c2;												// row/col coordinates of uniform ROI
	long NeROI;														// ROI nbr of elements

    // surrounding pixel indicies
    int *iN,*iS,*jE,*jW;    

    // counters
    int iter;   // primary loop
    long i,j;    // image row/col

	// memory sizes
	int mem_size_i;
	int mem_size_j;
	int mem_size_single;

	//================================================================================80
	// 	GPU VARIABLES
	//================================================================================80

	// CUDA kernel execution parameters
	dim3 threads;
	int blocks_x;
	dim3 blocks;
	dim3 blocks2;
	dim3 blocks3;

	// memory sizes
	int mem_size;															// matrix memory size

	// HOST
	int no;
	int mul;
	fp total;
	fp total2;
	fp meanROI;
	fp meanROI2;
	fp varROI;
	fp q0sqr;

	// DEVICE
	fp* sums;															// partial sum
	fp* sums2;
	int* tmp_iN;
	int* tmp_iS;
	int* tmp_jE;
	int* tmp_jW;
	fp* dN; 
	fp* dS; 
	fp* dW; 
	fp* dE;
	fp* I;																// input IMAGE on DEVICE
	fp* c;

	time1 = get_time();

	//================================================================================80
	// 	GET INPUT PARAMETERS
	//================================================================================80

	if(argc != 6){
		printf("ERROR: wrong number of arguments\n");
		return 0;
	}
	else{
		niter = atoi(argv[1]);
		lambda = atof(argv[2]);
		Nr = atoi(argv[3]);						// it is 502 in the original image
		Nc = atoi(argv[4]);						// it is 458 in the original image
	}

	time2 = get_time();

	//================================================================================80
	// 	READ IMAGE (SIZE OF IMAGE HAS TO BE KNOWN)
	//================================================================================80
	
	Ne = Nr*Nc;

    // read image
	image_ori_rows = 502;
	image_ori_cols = 458;
	image_ori_elem = image_ori_rows * image_ori_cols;

	tmp_image_ori = (fp*)malloc(sizeof(fp) * image_ori_elem);
	tmp_image = (fp*)malloc(sizeof(fp) * Ne);

	read_graphics(	"../../../data/srad/image.pgm",
								tmp_image_ori,
								image_ori_rows,
								image_ori_cols,
								1);

	// resize(	tmp_image_ori,
	// 	image_ori_rows,
	// 	image_ori_cols,
	// 	tmp_image,
	// 	Nr,
	// 	Nc,
	// 	1);

	r1     = 0;											// top row index of ROI
    r2     = Nr - 1;									// bottom row index of ROI
    c1     = 0;											// left column index of ROI
    c2     = Nc - 1;									// right column index of ROI

	// ROI image size
	NeROI = (r2-r1+1)*(c2-c1+1);											// number of elements in ROI, ROI size

	// allocate variables for surrounding pixels
	mem_size = sizeof(fp) * Ne;
	mem_size_i = sizeof(int) * Nr;											//
	mem_size_j = sizeof(int) * Nc;

	size_t membytes = 0;
	membytes += page_align(mem_size);
	membytes += page_align(mem_size_i);
	// membytes += page_align(mem_size_i);
	// membytes += page_align(mem_size_j);
	// membytes += page_align(mem_size_j);
	membytes += page_align(mem_size);
	membytes += page_align(mem_size);
	membytes += page_align(mem_size);
	membytes += page_align(mem_size);
	membytes += page_align(mem_size);
	membytes += page_align(mem_size);
	membytes += page_align(mem_size);
  	membytes += 309 * 1024 * 1024;

	threads.x = NUMBER_THREADS;												// define the number of threads in the block
	threads.y = 1;
	blocks_x = Ne/threads.x;
	if (Ne % threads.x != 0){												// compensate for division remainder above by adding one grid
		blocks_x = blocks_x + 1;																	
	}
	blocks.x = blocks_x;													// define the number of blocks in the grid
	blocks.y = 1;

	// printf("blocks_x: %d, blocks_y: %d, threads_x: %d, threads_y:%d\n", blocks.x, blocks.y, threads.x, threads.y);

    clock_gettime( CLOCK_REALTIME, &specific_time);
    now = localtime(&specific_time.tv_sec);
    millsec = specific_time.tv_nsec;

    millsec = floor (specific_time.tv_nsec/1.0e6);

    printf("TID: %d before schedule, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
		now->tm_mon + 1, now->tm_mday, now->tm_hour, 
		now->tm_min, now->tm_sec, millsec);

	long orig_alloc_mem = bemps_begin(tid, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z, membytes, ret_dev_id);

	clock_gettime( CLOCK_REALTIME, &specific_time);
    now = localtime(&specific_time.tv_sec);
    millsec = specific_time.tv_nsec;

    millsec = floor (specific_time.tv_nsec/1.0e6);

    printf("TID: %d after schedule, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
        now->tm_mon + 1, now->tm_mday, now->tm_hour, 
        now->tm_min, now->tm_sec, millsec);
    
    if (membytes <= orig_alloc_mem){
		full = 1;
		priority = -5;
	}
	else{
		full = 0;
		priority = 0;
	}

	printf("Full: %d\n", full);

    printf("ret_dev_id: %d\n", ret_dev_id);

	// int low, max;

	// CUDA_CHECK(hipDeviceGetStreamPriorityRange(&low, &max));

	// printf("Low: %d, Max: %d\n", low, max);

	// int* dummy;
	// size_t d_size = 13127122944;
	// CUDA_CHECK(hipMalloc(&dummy, d_size));
	// CUDA_CHECK(hipMemset(dummy, 0, d_size));

	// t_start = high_resolution_clock::now();

	// hipMallocManaged(&image_ori, sizeof(fp) * image_ori_elem);

	// t_stop = high_resolution_clock::now();
	// duration = duration_cast<milliseconds>(t_stop - t_start);
	// std::cout << "Tid: " << tid << " First device mem allocation time: " << duration.count() << std::endl;

	time3 = get_time();

	//================================================================================80
	// 	KERNEL EXECUTION PARAMETERS
	//================================================================================80

	time4 = get_time();

	//================================================================================80
	// 	RESIZE IMAGE (ASSUMING COLUMN MAJOR STORAGE OF image_orig)
	//================================================================================80

	// image = (fp*)malloc(sizeof(fp) * Ne);

	auto t_start = high_resolution_clock::now();

	CUDA_CHECK(hipMallocManaged(&image, sizeof(fp) * Ne));

	CUDA_CHECK(hipMallocManaged(&iN, mem_size_i));
	CUDA_CHECK(hipMallocManaged(&iS, mem_size_i));

	CUDA_CHECK(hipMallocManaged(&jW, mem_size_j));
	CUDA_CHECK(hipMallocManaged(&jE, mem_size_j));

	CUDA_CHECK(hipMallocManaged(&sums, mem_size));
	CUDA_CHECK(hipMallocManaged(&sums2, mem_size));
	CUDA_CHECK(hipMallocManaged(&dN, mem_size));
	CUDA_CHECK(hipMallocManaged(&dS, mem_size));
	CUDA_CHECK(hipMallocManaged(&dW, mem_size));
	CUDA_CHECK(hipMallocManaged(&dE, mem_size));
	CUDA_CHECK(hipMallocManaged(&c, mem_size));

	auto t_stop = high_resolution_clock::now();
	auto duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Device mem allocation time: " << duration.count() << std::endl;

	// resize(	image_ori,
	// 			image_ori_rows,
	// 			image_ori_cols,
	// 			image,
	// 			Nr,
	// 			Nc,
	// 			1);

	ret1.devPtr = image;
	ret1.advice = hipMemAdviseSetPreferredLocation;
	ret1.device = ret_dev_id;
	ret1.alloc_size = mem_size;

	ret2.devPtr = iN;
	ret2.advice = hipMemAdviseSetPreferredLocation;
	ret2.device = ret_dev_id;
	ret2.alloc_size = mem_size_i;

	ret3.devPtr = iS;
	ret3.advice = hipMemAdviseSetPreferredLocation;
	ret3.device = ret_dev_id;
	ret3.alloc_size = mem_size_i;
	
	ret4.devPtr = jW;
	ret4.advice = hipMemAdviseSetPreferredLocation;
	ret4.device = ret_dev_id;
	ret4.alloc_size = mem_size_j;

	ret5.devPtr = jE;
	ret5.advice = hipMemAdviseSetPreferredLocation;
	ret5.device = ret_dev_id;
	ret5.alloc_size = mem_size_j;

	ret6.devPtr = sums;
	ret6.advice = hipMemAdviseSetPreferredLocation;
	ret6.device = ret_dev_id;
	ret6.alloc_size = mem_size;

	ret7.devPtr = sums2;
	ret7.advice = hipMemAdviseSetPreferredLocation;
	ret7.device = ret_dev_id;
	ret7.alloc_size = mem_size;
	
	ret8.devPtr = dN;
	ret8.advice = hipMemAdviseSetPreferredLocation;
	ret8.device = ret_dev_id;
	ret8.alloc_size = mem_size;

	ret9.devPtr = dS;
	ret9.advice = hipMemAdviseSetPreferredLocation;
	ret9.device = ret_dev_id;
	ret9.alloc_size = mem_size;

	ret10.devPtr = dW;
	ret10.advice = hipMemAdviseSetPreferredLocation;
	ret10.device = ret_dev_id;
	ret10.alloc_size = mem_size;

	ret11.devPtr = dE;
	ret11.advice = hipMemAdviseSetPreferredLocation;
	ret11.device = ret_dev_id;
	ret11.alloc_size = mem_size;

	ret12.devPtr = c;
	ret12.advice = hipMemAdviseSetPreferredLocation;
	ret12.device = ret_dev_id;
	ret12.alloc_size = mem_size;

	mem_list.push_back(ret1);
	mem_list.push_back(ret2);
	mem_list.push_back(ret3);
	mem_list.push_back(ret4);
	mem_list.push_back(ret5);
	mem_list.push_back(ret6);
	mem_list.push_back(ret7);
	mem_list.push_back(ret8);
	mem_list.push_back(ret9);
	mem_list.push_back(ret10);
	mem_list.push_back(ret11);
	mem_list.push_back(ret12);

	time5 = get_time();

	//================================================================================80
	// 	SETUP
	//================================================================================80

	t_start = high_resolution_clock::now();

	int fd = -1;
	ssize_t ret = -1;

	if(Nr == 11000){
		printf("Small\n");
		fd = open("small/iN_small.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, iN, mem_size_i);
		printf("%zd\n", ret);
		close(fd);

		fd = open("small/iS_small.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, iS, mem_size_i);
		printf("%zd\n", ret);
		close(fd);

		fd = open("small/jW_small.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, jW, mem_size_j);
		printf("%zd\n", ret);
		close(fd);

		fd = open("small/jE_small.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, jE, mem_size_j);
		printf("%zd\n", ret);
		close(fd);
	}
	else if(Nr == 15000){
		printf("Large1\n");
		fd = open("large1/iN_large1.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, iN, mem_size_i);
		printf("%zd\n", ret);
		close(fd);

		fd = open("large1/iS_large1.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, iS, mem_size_i);
		printf("%zd\n", ret);
		close(fd);

		fd = open("large1/jW_large1.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, jW, mem_size_j);
		printf("%zd\n", ret);
		close(fd);

		fd = open("large1/jE_large1.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, jE, mem_size_j);
		printf("%zd\n", ret);
		close(fd);
	}
	else if(Nr = 20000){
		printf("Large2\n");
		fd = open("large2/iN_large2.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, iN, mem_size_i);
		printf("%zd\n", ret);
		close(fd);

		fd = open("large2/iS_large2.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, iS, mem_size_i);
		printf("%zd\n", ret);
		close(fd);

		fd = open("large2/jW_large2.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, jW, mem_size_j);
		printf("%zd\n", ret);
		close(fd);

		fd = open("large2/jE_large2.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, jE, mem_size_j);
		printf("%zd\n", ret);
		close(fd);
	}

	resize(	tmp_image_ori,
		image_ori_rows,
		image_ori_cols,
		image,
		Nr,
		Nc,
		1);

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Host initialization time: " << duration.count() << std::endl;

	t_start = high_resolution_clock::now();

	if(full == 1){
		for(Parameters var : mem_list){
			// if(loop < 5){
			// 	CUDA_CHECK(hipMemPrefetchAsync(ret.devPtr, ret.alloc_size, ret.device, 0));
			// }
			CUDA_CHECK(hipMemAdvise(var.devPtr, var.alloc_size, var.advice, var.device));
		}
		CUDA_CHECK(hipMemPrefetchAsync(image, mem_size, ret_dev_id, 0));
		CUDA_CHECK(hipMemPrefetchAsync(iN, mem_size_i, ret_dev_id, 0));
		CUDA_CHECK(hipMemPrefetchAsync(iS, mem_size_i, ret_dev_id, 0));
		CUDA_CHECK(hipMemPrefetchAsync(jW, mem_size_j, ret_dev_id, 0));
		CUDA_CHECK(hipMemPrefetchAsync(jE, mem_size_j, ret_dev_id, 0));
	}

	// size_t free_mem, total_mem;
	// CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

	// printf("Free: %zd, Total: %zd\n", free_mem, total_mem);

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Host to Device memcpy time: " << duration.count() << std::endl;

	t_start = high_resolution_clock::now();

	// if(full == 1){
	// 	int loop = 0;
	// 	printf("Hi\n");
	// 	for(Parameters ret : mem_list){
	// 		// if(loop < 5){
	// 		// 	CUDA_CHECK(hipMemPrefetchAsync(ret.devPtr, ret.alloc_size, ret.device, 0));
	// 		// }
	// 		CUDA_CHECK(hipMemAdvise(ret.devPtr, ret.alloc_size, ret.advice, ret.device));
	// 		loop++;
	// 	}
	// }

	// CUDA_CHECK(hipDeviceSynchronize());

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Mem advise time: " << duration.count() << std::endl;

	//================================================================================80
	// 	GPU SETUP
	//================================================================================80

	// hipMemPrefetchAsync(iN, mem_size_i, 0, 0);
	// hipMemPrefetchAsync(iS, mem_size_i, 0, 0);
	// hipMemPrefetchAsync(jE, mem_size_j, 0, 0);
	// hipMemPrefetchAsync(jW, mem_size_j, 0, 0);

	// checkCUDAError("setup");

	//================================================================================80
	// 	COPY INPUT TO CPU
	//================================================================================80

	// hipMemPrefetchAsync(image, mem_size, ret_dev_id, 0);

	time6 = get_time();

	if(full == 0){
		clock_gettime( CLOCK_REALTIME, &specific_time);
		now = localtime(&specific_time.tv_sec);
		millsec = specific_time.tv_nsec;

		millsec = floor (specific_time.tv_nsec/1.0e6);

		printf("TID: %d waiting start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
			now->tm_mon + 1, now->tm_mday, now->tm_hour, 
			now->tm_min, now->tm_sec, millsec);

		el_wait(tid);

		clock_gettime( CLOCK_REALTIME, &specific_time);
		now = localtime(&specific_time.tv_sec);
		millsec = specific_time.tv_nsec;

		millsec = floor (specific_time.tv_nsec/1.0e6);

		printf("TID: %d waiting end, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
			now->tm_mon + 1, now->tm_mday, now->tm_hour, 
			now->tm_min, now->tm_sec, millsec);
	}

	//================================================================================80
	// 	SCALE IMAGE DOWN FROM 0-255 TO 0-1 AND EXTRACT
	//================================================================================80

	hipEvent_t e_start, e_stop;
	CUDA_CHECK(hipEventCreate(&e_start));
	CUDA_CHECK(hipEventCreate(&e_stop));

	float total_e, time;

	// t_start = high_resolution_clock::now();

	hipStream_t s1;
	CUDA_CHECK(hipStreamCreateWithPriority(&s1, 0, priority));

	hipEvent_t event1;
	CUDA_CHECK(hipEventCreateWithFlags(&event1, hipEventDisableTiming));

	CUDA_CHECK(hipEventRecord(e_start));

	// if(full == 1){
	// 	clock_gettime( CLOCK_REALTIME, &specific_time);
	// 			now = localtime(&specific_time.tv_sec);
	// 			millsec = specific_time.tv_nsec;

	// 			millsec = floor (specific_time.tv_nsec/1.0e6);

	// 			printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
	// 				now->tm_mon + 1, now->tm_mday, now->tm_hour, 
	// 				now->tm_min, now->tm_sec, millsec);

	// 			nl_signal(tid);

	// 			clock_gettime( CLOCK_REALTIME, &specific_time);
	// 			now = localtime(&specific_time.tv_sec);
	// 			millsec = specific_time.tv_nsec;

	// 			millsec = floor (specific_time.tv_nsec/1.0e6);

	// 			printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
	// 				now->tm_mon + 1, now->tm_mday, now->tm_hour, 
	// 				now->tm_min, now->tm_sec, millsec);
	// }
	
	extract<<<blocks, threads, 0, s1>>>(	Ne,
									image);

	// extract<<<blocks, threads>>>(	Ne,
	// 	image);

	CUDA_CHECK(hipEventRecord(event1, s1));

	task_monitoring(event1, tid, orig_alloc_mem, membytes);

	CUDA_CHECK(hipEventRecord(e_stop));
	CUDA_CHECK(hipEventSynchronize(e_stop));
	CUDA_CHECK(hipEventElapsedTime(&total_e, e_start, e_stop));

	// CUDA_CHECK(hipDeviceSynchronize());

	printf("E_kernel: %f\n", total_e);

	if(full == 1){
		clock_gettime( CLOCK_REALTIME, &specific_time);
				now = localtime(&specific_time.tv_sec);
				millsec = specific_time.tv_nsec;

				millsec = floor (specific_time.tv_nsec/1.0e6);

				printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
					now->tm_mon + 1, now->tm_mday, now->tm_hour, 
					now->tm_min, now->tm_sec, millsec);

				nl_signal(tid);

				clock_gettime( CLOCK_REALTIME, &specific_time);
				now = localtime(&specific_time.tv_sec);
				millsec = specific_time.tv_nsec;

				millsec = floor (specific_time.tv_nsec/1.0e6);

				printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
					now->tm_mon + 1, now->tm_mday, now->tm_hour, 
					now->tm_min, now->tm_sec, millsec);
	}
	
	// t_stop = high_resolution_clock::now();
	// duration = duration_cast<milliseconds>(t_stop - t_start);
	// std::cout << "Tid: " << tid << " Kernel execution time: " << duration.count() << std::endl;

	// checkCUDAError("extract");

	time7 = get_time();

	//================================================================================80
	// 	COMPUTATION
	//================================================================================80

	hipEvent_t *p_start, *p_stop, *r_start, *r_stop, *s1_start, *s1_stop, *s2_start, *s2_stop, *cm_start, *cm_stop;

	// CUDA_CHECK(hipMallocManaged((void **)&p_start, sizeof(hipEvent_t) * 100))
	// CUDA_CHECK(hipMallocManaged((void **)&p_stop, sizeof(hipEvent_t) * 100))

	p_start = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 100);
	p_stop = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 100);
	r_start = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 400);
	r_stop = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 400);
	s1_start = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 100);
	s1_stop = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 100);
	s2_start = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 100);
	s2_stop = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 100);
	cm_start = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 100);
	cm_stop = (hipEvent_t *)malloc(sizeof(hipEvent_t) * 100);

	// CUDA_CHECK(hipEventCreate(p_start));
	// CUDA_CHECK(hipEventCreate(p_stop));
	// hipEventCreate(&r_start);
	// hipEventCreate(&r_stop);
	// hipEventCreate(&s1_start);
	// hipEventCreate(&s1_stop);
	// hipEventCreate(&s2_start);
	// hipEventCreate(&s2_stop);

	float total_p, total_r, total_s1, total_s2, memcpy_time;
	total_p = 0;
	total_r = 0;
	total_s1 = 0;
	total_s2 = 0;
	memcpy_time = 0;

	// printf("iterations: ");
	hipStream_t s2, s3;
	hipEvent_t event2, event3;
	// execute main loop

	t_start = high_resolution_clock::now();

	for (iter=0; iter<niter; iter++){										// do for the number of iterations input parameter
		int r_iter = 0;
	// printf("%d ", iter);
	// fflush(NULL);

		// if(iter == 0){
		// 	CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

		// 	printf("Free: %zd, Total: %zd\n", free_mem, total_mem);
		// }

		CUDA_CHECK(hipStreamCreateWithPriority(&s2, 0, priority));

		CUDA_CHECK(hipEventCreateWithFlags(&event2, hipEventDisableTiming));

		CUDA_CHECK(hipEventCreate(&p_start[iter]));
		CUDA_CHECK(hipEventCreate(&p_stop[iter]));
		CUDA_CHECK(hipEventRecord(p_start[iter]));

		// execute square kernel
		prepare<<<blocks, threads, 0, s2>>>(	Ne,
										image,
										sums,
										sums2);

		// prepare<<<blocks, threads>>>(	Ne,
		// 	image,
		// 	sums,
		// 	sums2);

		CUDA_CHECK(hipEventRecord(event2, s2));

		task_monitoring(event2, tid, orig_alloc_mem, membytes);
								
		CUDA_CHECK(hipEventRecord(p_stop[iter]));
		// hipEventSynchronize(p_stop);
		// hipEventElapsedTime(&time, p_start, p_stop);
		// total_p += time;

		// CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipStreamSynchronize(s2));

		CUDA_CHECK(hipStreamDestroy(s2));
		CUDA_CHECK(hipEventDestroy(event2));

		// checkCUDAError("prepare");

		// performs subsequent reductions of sums
		blocks2.x = blocks.x;												// original number of blocks
		blocks2.y = blocks.y;												
		no = Ne;														// original number of sum elements
		mul = 1;														// original multiplier
		
		// CUDA_CHECK(hipEventCreate(&r_start[iter]));
		// CUDA_CHECK(hipEventCreate(&r_stop[iter]));
		// CUDA_CHECK(hipEventRecord(r_start[iter]));

		// if(iter == 0){
		// 	CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

		// 	printf("Free: %zd, Total: %zd\n", free_mem, total_mem);
		// }

		while(blocks2.x != 0){
			// printf("Hello\n");
			checkCUDAError("before reduce");

			CUDA_CHECK(hipStreamCreateWithPriority(&s3, 0, priority));

			CUDA_CHECK(hipEventCreateWithFlags(&event3, hipEventDisableTiming));

			CUDA_CHECK(hipEventCreate(&r_start[iter*4+r_iter]));
			CUDA_CHECK(hipEventCreate(&r_stop[iter*4+r_iter]));
			CUDA_CHECK(hipEventRecord(r_start[iter*4+r_iter]));


			// hipEventRecord(r_start);

			// run kernel
			reduce<<<blocks2, threads, 0, s3>>>(	Ne,
											no,
											mul,
											sums, 
											sums2);

			// reduce<<<blocks2, threads>>>(	Ne,
			// 	no,
			// 	mul,
			// 	sums, 
			// 	sums2);

			CUDA_CHECK(hipEventRecord(event3, s3));

			task_monitoring(event3, tid, orig_alloc_mem, membytes);
			
			CUDA_CHECK(hipEventRecord(r_stop[iter*4+r_iter]));

			// hipEventRecord(r_stop);
			// hipEventSynchronize(r_stop);
			// hipEventElapsedTime(&time, r_start, r_stop);
			// total_r += time;

			// CUDA_CHECK(hipDeviceSynchronize());
			CUDA_CHECK(hipStreamSynchronize(s3));
									
			CUDA_CHECK(hipStreamDestroy(s3));
			CUDA_CHECK(hipEventDestroy(event3));

			// checkCUDAError("reduce");

			// update execution parameters
			no = blocks2.x;												// get current number of elements
			if(blocks2.x == 1){
				blocks2.x = 0;
			}
			else{
				mul = mul * NUMBER_THREADS;									// update the increment
				blocks_x = blocks2.x/threads.x;								// number of blocks
				if (blocks2.x % threads.x != 0){							// compensate for division remainder above by adding one grid
					blocks_x = blocks_x + 1;
				}
				blocks2.x = blocks_x;
				blocks2.y = 1;
			}

			checkCUDAError("after reduce");
			
			r_iter++;
		}

		// CUDA_CHECK(hipEventRecord(r_stop[iter]));

		checkCUDAError("before copy sum");

		CUDA_CHECK(hipEventCreate(&cm_start[iter]));
		CUDA_CHECK(hipEventCreate(&cm_stop[iter]));
		CUDA_CHECK(hipEventRecord(cm_start[iter]));
		
		// copy total sums to device
		mem_size_single = sizeof(fp) * 1;
		// hipMemcpy(&total, sums, mem_size_single, hipMemcpyDeviceToHost);
		total = sums[0];
		// CUDA_CHECK(hipMemPrefetchAsync(sums, mem_size_single, hipCpuDeviceId, 0));
		// hipMemcpy(&total2, sums2, mem_size_single, hipMemcpyDeviceToHost);
		total2 = sums2[0];
		// CUDA_CHECK(hipMemPrefetchAsync(sums2, mem_size_single, hipCpuDeviceId, 0));

		checkCUDAError("copy sum");

		CUDA_CHECK(hipEventRecord(cm_stop[iter]));

		// auto cp_start = high_resolution_clock::now();

		// calculate statistics
		meanROI	= total / fp(NeROI);										// gets mean (average) value of element in ROI
		meanROI2 = meanROI * meanROI;										//
		varROI = (total2 / fp(NeROI)) - meanROI2;						// gets variance of ROI								
		q0sqr = varROI / meanROI2;											// gets standard deviation of ROI

		// auto cp_stop = high_resolution_clock::now();
		// auto cp_duration = duration_cast<milliseconds>(cp_stop - cp_start);
		// std::cout << "Host Processing: " << cp_duration.count() << std::endl;
		// memcpy_time += cp_duration.count();

		hipStream_t s4;
		CUDA_CHECK(hipStreamCreateWithPriority(&s4, 0, priority));

		hipEvent_t event4;
		CUDA_CHECK(hipEventCreateWithFlags(&event4, hipEventDisableTiming));

		CUDA_CHECK(hipEventCreate(&s1_start[iter]));
		CUDA_CHECK(hipEventCreate(&s1_stop[iter]));
		CUDA_CHECK(hipEventRecord(s1_start[iter]));

		// execute srad kernel
		srad<<<blocks, threads, 0, s4>>>(	lambda,									// SRAD coefficient 
									Nr,										// # of rows in input image
									Nc,										// # of columns in input image
									Ne,										// # of elements in input image
									iN,									// indices of North surrounding pixels
									iS,									// indices of South surrounding pixels
									jE,									// indices of East surrounding pixels
									jW,									// indices of West surrounding pixels
									dN,									// North derivative
									dS,									// South derivative
									dW,									// West derivative
									dE,									// East derivative
									q0sqr,									// standard deviation of ROI 
									c,									// diffusion coefficient
									image);									// output image

		// srad<<<blocks, threads>>>(	lambda,									// SRAD coefficient 
		// 	Nr,										// # of rows in input image
		// 	Nc,										// # of columns in input image
		// 	Ne,										// # of elements in input image
		// 	iN,									// indices of North surrounding pixels
		// 	iS,									// indices of South surrounding pixels
		// 	jE,									// indices of East surrounding pixels
		// 	jW,									// indices of West surrounding pixels
		// 	dN,									// North derivative
		// 	dS,									// South derivative
		// 	dW,									// West derivative
		// 	dE,									// East derivative
		// 	q0sqr,									// standard deviation of ROI 
		// 	c,									// diffusion coefficient
		// 	image);									// output image

		CUDA_CHECK(hipEventRecord(event4, s4));

		task_monitoring(event4, tid, orig_alloc_mem, membytes);
				
		CUDA_CHECK(hipEventRecord(s1_stop[iter]));
		// hipEventSynchronize(s1_stop);
		// hipEventElapsedTime(&time, s1_start, s1_stop);
		// total_s1 += time;

		// CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipStreamSynchronize(s4));

		CUDA_CHECK(hipStreamDestroy(s4));
		CUDA_CHECK(hipEventDestroy(event4));

		// checkCUDAError("srad");

		hipStream_t s5;
		CUDA_CHECK(hipStreamCreateWithPriority(&s5, 0, priority));

		hipEvent_t event5;
		CUDA_CHECK(hipEventCreateWithFlags(&event5, hipEventDisableTiming));

		CUDA_CHECK(hipEventCreate(&s2_start[iter]));
		CUDA_CHECK(hipEventCreate(&s2_stop[iter]));
		CUDA_CHECK(hipEventRecord(s2_start[iter]));

		// execute srad2 kernel
		srad2<<<blocks, threads, 0, s5>>>(	lambda,									// SRAD coefficient 
									Nr,										// # of rows in input image
									Nc,										// # of columns in input image
									Ne,										// # of elements in input image
									iN,									// indices of North surrounding pixels
									iS,									// indices of South surrounding pixels
									jE,									// indices of East surrounding pixels
									jW,									// indices of West surrounding pixels
									dN,									// North derivative
									dS,									// South derivative
									dW,									// West derivative
									dE,									// East derivative
									c,									// diffusion coefficient
									image);									// output image

		// srad2<<<blocks, threads>>>(	lambda,									// SRAD coefficient 
		// 	Nr,										// # of rows in input image
		// 	Nc,										// # of columns in input image
		// 	Ne,										// # of elements in input image
		// 	iN,									// indices of North surrounding pixels
		// 	iS,									// indices of South surrounding pixels
		// 	jE,									// indices of East surrounding pixels
		// 	jW,									// indices of West surrounding pixels
		// 	dN,									// North derivative
		// 	dS,									// South derivative
		// 	dW,									// West derivative
		// 	dE,									// East derivative
		// 	c,									// diffusion coefficient
		// 	image);									// output image

		CUDA_CHECK(hipEventRecord(event5, s5));

		task_monitoring(event5, tid, orig_alloc_mem, membytes);
							
		CUDA_CHECK(hipEventRecord(s2_stop[iter]));
		// hipEventSynchronize(s2_stop);
		// hipEventElapsedTime(&time, s2_start, s2_stop);
		// total_s2 += time;

		// CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipStreamSynchronize(s5));
		
		CUDA_CHECK(hipStreamDestroy(s5));
		CUDA_CHECK(hipEventDestroy(event5));

		// checkCUDAError("srad2");

	}

	CUDA_CHECK(hipEventSynchronize(p_stop[99]));
	CUDA_CHECK(hipEventSynchronize(r_stop[399]));
	CUDA_CHECK(hipEventSynchronize(s1_stop[99]));
	CUDA_CHECK(hipEventSynchronize(s2_stop[99]));

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Kernel execution time: " << duration.count() << std::endl;

	// CUDA_CHECK(hipEventSynchronize(p_stop[99]));
	// CUDA_CHECK(hipEventSynchronize(r_stop[99]));
	// CUDA_CHECK(hipEventSynchronize(s1_stop[99]));
	// CUDA_CHECK(hipEventSynchronize(s2_stop[99]));

	for(int i = 0; i < 100; i++){
		hipEventElapsedTime(&time, p_start[i], p_stop[i]);
		// printf("%f ", time);
		total_p += time;
		for(int j = 0; j < 4; j++){
			hipEventElapsedTime(&time, r_start[i*4+j], r_stop[i*4+j]);
			// printf("%f ", time);
			total_r += time;
		}
		hipEventElapsedTime(&time, cm_start[i], cm_stop[i]);
		// printf("%f ", time);
		memcpy_time += time;
		hipEventElapsedTime(&time, s1_start[i], s1_stop[i]);
		// printf("%f ", time);
		total_s1 += time;
		hipEventElapsedTime(&time, s2_start[i], s2_stop[i]);
		// printf("%f\n", time);
		total_s2 += time;
	}
	


	printf("P_kernel: %f, R_kernel: %f, Memcpy: %f, S1_kernel: %f, S2_kernel %f\n", total_p, total_r,  memcpy_time, total_s1, total_s2);
	
	// printf("\n");

	time8 = get_time();

	t_start = high_resolution_clock::now();

	hipFree(c);
	hipFree(iN);
	hipFree(iS);
	hipFree(jE);
	hipFree(jW);
	hipFree(dN);
	hipFree(dS);
	hipFree(dE);
	hipFree(dW);
	hipFree(sums);
	hipFree(sums2);

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Memory deallocation time: " << duration.count() << std::endl;

	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);
	mem_list.erase(mem_list.begin()+1);

	if(full == 1){
		ef_mem = page_align(page_align(mem_size_i) + page_align(mem_size) * 7);
		pre_bemps_free(tid, ef_mem);
		ef_cnt = 1;
	}
	
	//================================================================================80
	// 	SCALE IMAGE UP FROM 0-1 TO 0-255 AND COMPRESS
	//================================================================================80

	// hipFree(c);
	// hipFree(iN);
	// hipFree(iS);
	// hipFree(jE);
	// hipFree(jW);
	// hipFree(dN);
	// hipFree(dS);
	// hipFree(dE);
	// hipFree(dW);
	// hipFree(sums);
	// hipFree(sums2);

	// int64_t eager_free_memory = (int64_t) 7 * (int64_t) mem_size + (int64_t) 2 * (int64_t) mem_size_i + (int64_t) 2 * (int64_t) mem_size_j;
	
	// if(full == 1){
	// 	pre_bemps_free(tid, eager_free_memory);
	// 	ef_cnt = 1;
	// }

	// t_start = high_resolution_clock::now();

	hipEvent_t c_start, c_stop;
	CUDA_CHECK(hipEventCreate(&c_start));
	CUDA_CHECK(hipEventCreate(&c_stop));

	float total_c;

	hipStream_t s6;
	CUDA_CHECK(hipStreamCreateWithPriority(&s6, 0, priority));

	hipEvent_t event6;
	CUDA_CHECK(hipEventCreateWithFlags(&event6, hipEventDisableTiming));

	CUDA_CHECK(hipEventRecord(c_start));

	compress<<<blocks, threads, 0, s6>>>(	Ne,
									image);

	// compress<<<blocks, threads>>>(	Ne,
	// 	image);

	// if(full == 1){
	// 	nl_signal(tid);
	// }

	CUDA_CHECK(hipEventRecord(event6, s6));

	task_monitoring(event6, tid, orig_alloc_mem, membytes);
								
	// CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipStreamDestroy(s6));
	CUDA_CHECK(hipEventDestroy(event6));

	CUDA_CHECK(hipEventRecord(c_stop));
	CUDA_CHECK(hipEventSynchronize(c_stop));
	CUDA_CHECK(hipEventElapsedTime(&total_c, c_start, c_stop));

	printf("C_kernel: %f\n", total_c);
	// t_stop = high_resolution_clock::now();
	// duration = duration_cast<milliseconds>(t_stop - t_start);
	// std::cout << "Tid: " << tid << " Kernel execution time: " << duration.count() << std::endl;

	// checkCUDAError("compress");

	time9 = get_time();

	//================================================================================80
	// 	COPY RESULTS BACK TO CPU
	//================================================================================80

	t_start = high_resolution_clock::now();

	// hipMemcpy(tmp_image, image, mem_size, hipMemcpyDeviceToHost);
	CUDA_CHECK(hipMemPrefetchAsync(image, mem_size, hipCpuDeviceId, 0));

	checkCUDAError("copy back");

	// CUDA_CHECK(hipDeviceSynchronize());

	// memcpy(tmp_image, image, mem_size);

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Device to Host memcpy time: " << duration.count() << std::endl;

	time10 = get_time();

	t_start = high_resolution_clock::now();

	char* str1 = argv[5];
	char* str2 = "_nongds_output.txt";
	strcat(str1, str2);

	int o_fd = -1;
	o_fd = open(str1, O_CREAT | O_RDWR, 0664);
	if (o_fd < 0) {
		std::cerr << "file open error:" << std::endl;
	}

	printf("%d\n", o_fd);

	ssize_t o_ret = -1;
	o_ret = write(o_fd, image, mem_size);
	
	printf("%zd\n", o_ret);

	close(o_fd);

	// write_graphics(	"image_out.pgm",
	// 				tmp_image,
	// 				Nr,
	// 				Nc,
	// 				1,
	// 				255);

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Host to Storage memcpy time: " << duration.count() << std::endl;

	t_start = high_resolution_clock::now();

	// hipFree(image_ori);
	hipFree(image);

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Device memory deallocation time: " << duration.count() << std::endl;

	bemps_free(tid);

	clock_gettime( CLOCK_REALTIME, &specific_time);
    now = localtime(&specific_time.tv_sec);
    millsec = specific_time.tv_nsec;

    millsec = floor (specific_time.tv_nsec/1.0e6);

    printf("TID: %d finish work, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
        now->tm_mon + 1, now->tm_mday, now->tm_hour, 
        now->tm_min, now->tm_sec, millsec);
		
	//================================================================================80
	// 	WRITE IMAGE AFTER PROCESSING
	//================================================================================80

	time11 = get_time();

	//================================================================================80
	//	DEALLOCATE
	//================================================================================80

	// free(image_ori);
	// free(image);
	// free(iN); 
	// free(iS); 
	// free(jW); 
	// free(jE);

	time12 = get_time();

	//================================================================================80
	//	DISPLAY TIMING
	//================================================================================80

	printf("Time spent in different stages of the application:\n");
	printf("%15.12f s, %15.12f % : SETUP VARIABLES\n", 														(float) (time1-time0) / 1000000, (float) (time1-time0) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : READ COMMAND LINE PARAMETERS\n", 										(float) (time2-time1) / 1000000, (float) (time2-time1) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : READ IMAGE FROM FILE\n", 												(float) (time3-time2) / 1000000, (float) (time3-time2) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU DRIVER INIT, CPU/GPU SETUP, MEMORY ALLOCATION\n", 														(float) (time4-time3) / 1000000, (float) (time4-time3) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : RESIZE IMAGE\n", 					(float) (time5-time4) / 1000000, (float) (time5-time4) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : COPY DATA TO CPU->GPU\n", 												(float) (time6-time5) / 1000000, (float) (time6-time5) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : EXTRACT IMAGE\n", 														(float) (time7-time6) / 1000000, (float) (time7-time6) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : COMPUTE\n", 																(float) (time8-time7) / 1000000, (float) (time8-time7) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : COMPRESS IMAGE\n", 														(float) (time9-time8) / 1000000, (float) (time9-time8) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : COPY DATA TO GPU->CPU\n", 												(float) (time10-time9) / 1000000, (float) (time10-time9) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : SAVE IMAGE INTO FILE\n", 												(float) (time11-time10) / 1000000, (float) (time11-time10) / (float) (time12-time0) * 100);
	printf("%15.12f s, %15.12f % : FREE MEMORY\n", 															(float) (time12-time11) / 1000000, (float) (time12-time11) / (float) (time12-time0) * 100);
	printf("Total time:\n");
	printf("%.12f s\n", 																					(float) (time12-time0) / 1000000);

	clock_gettime( CLOCK_REALTIME, &specific_time);
	now = localtime(&specific_time.tv_sec);
	millsec = specific_time.tv_nsec;
  
	millsec = floor (specific_time.tv_nsec/1.0e6);
  
	printf("TID: %d Application end, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
		now->tm_mon + 1, now->tm_mday, now->tm_hour, 
		now->tm_min, now->tm_sec, millsec);
		
}

//====================================================================================================100
//	END OF FILE
//====================================================================================================100
