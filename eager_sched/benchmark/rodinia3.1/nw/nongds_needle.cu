#include "hip/hip_runtime.h"
#define LIMIT -999
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "needle.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <bemps.hpp>
#include <algorithm>
#include <vector>
#include <chrono>
#include <iostream>

#include "cufile.h"
#include <unistd.h>
#include <error.h>
#include <fcntl.h>
#include <errno.h>

using namespace std::chrono;

// includes, kernels
#include "needle_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);


int blosum62[24][24] = {
{ 4, -1, -2, -2,  0, -1, -1,  0, -2, -1, -1, -1, -1, -2, -1,  1,  0, -3, -2,  0, -2, -1,  0, -4},
{-1,  5,  0, -2, -3,  1,  0, -2,  0, -3, -2,  2, -1, -3, -2, -1, -1, -3, -2, -3, -1,  0, -1, -4},
{-2,  0,  6,  1, -3,  0,  0,  0,  1, -3, -3,  0, -2, -3, -2,  1,  0, -4, -2, -3,  3,  0, -1, -4},
{-2, -2,  1,  6, -3,  0,  2, -1, -1, -3, -4, -1, -3, -3, -1,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{ 0, -3, -3, -3,  9, -3, -4, -3, -3, -1, -1, -3, -1, -2, -3, -1, -1, -2, -2, -1, -3, -3, -2, -4},
{-1,  1,  0,  0, -3,  5,  2, -2,  0, -3, -2,  1,  0, -3, -1,  0, -1, -2, -1, -2,  0,  3, -1, -4},
{-1,  0,  0,  2, -4,  2,  5, -2,  0, -3, -3,  1, -2, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -2,  0, -1, -3, -2, -2,  6, -2, -4, -4, -2, -3, -3, -2,  0, -2, -2, -3, -3, -1, -2, -1, -4},
{-2,  0,  1, -1, -3,  0,  0, -2,  8, -3, -3, -1, -2, -1, -2, -1, -2, -2,  2, -3,  0,  0, -1, -4},
{-1, -3, -3, -3, -1, -3, -3, -4, -3,  4,  2, -3,  1,  0, -3, -2, -1, -3, -1,  3, -3, -3, -1, -4},
{-1, -2, -3, -4, -1, -2, -3, -4, -3,  2,  4, -2,  2,  0, -3, -2, -1, -2, -1,  1, -4, -3, -1, -4},
{-1,  2,  0, -1, -3,  1,  1, -2, -1, -3, -2,  5, -1, -3, -1,  0, -1, -3, -2, -2,  0,  1, -1, -4},
{-1, -1, -2, -3, -1,  0, -2, -3, -2,  1,  2, -1,  5,  0, -2, -1, -1, -1, -1,  1, -3, -1, -1, -4},
{-2, -3, -3, -3, -2, -3, -3, -3, -1,  0,  0, -3,  0,  6, -4, -2, -2,  1,  3, -1, -3, -3, -1, -4},
{-1, -2, -2, -1, -3, -1, -1, -2, -2, -3, -3, -1, -2, -4,  7, -1, -1, -4, -3, -2, -2, -1, -2, -4},
{ 1, -1,  1,  0, -1,  0,  0,  0, -1, -2, -2,  0, -1, -2, -1,  4,  1, -3, -2, -2,  0,  0,  0, -4},
{ 0, -1,  0, -1, -1, -1, -1, -2, -2, -1, -1, -1, -1, -2, -1,  1,  5, -2, -2,  0, -1, -1,  0, -4},
{-3, -3, -4, -4, -2, -2, -3, -2, -2, -3, -2, -3, -1,  1, -4, -3, -2, 11,  2, -3, -4, -3, -2, -4},
{-2, -2, -2, -3, -2, -1, -2, -3,  2, -1, -1, -2, -1,  3, -3, -2, -2,  2,  7, -1, -3, -2, -1, -4},
{ 0, -3, -3, -3, -1, -2, -2, -3, -3,  3,  1, -2,  1, -1, -2, -2,  0, -3, -1,  4, -3, -2, -1, -4},
{-2, -1,  3,  4, -3,  0,  1, -1,  0, -3, -4,  0, -3, -3, -2,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{-1,  0,  0,  1, -3,  3,  4, -2,  0, -3, -3,  1, -1, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -1, -1, -1, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -2,  0,  0, -2, -1, -1, -1, -1, -1, -4},
{-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,  1}
};

double gettime() {
  struct timeval t;
  gettimeofday(&t,NULL);
  return t.tv_sec+t.tv_usec*1e-6;
}

bool full; // 1 = fully secured
int priority;

struct Parameters{
	void* devPtr;
	size_t count;
	hipMemoryAdvise advice;
	int device;
	size_t alloc_size; // 디바이스 메모리에 올릴 페이지 크기
	// std::bitset<kernel_num> bit; // liveness check
};

std::vector<Parameters> mem_list;

#define CUDA_CHECK(val) { \
	if (val != hipSuccess) { \
		fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(val), __LINE__, __FILE__); \
		exit(val); \
	} \
}

size_t page_align (size_t mem){
	if((mem % 2097152) != 0){
		return (2097152 * (mem / 2097152 + 1));
	}
	else{
		return mem;
	}
}

void task_monitoring(hipEvent_t event, int tid, long orig_alloc_mem, size_t membytes){
	long update_mem = 0;
	long tmp_mem = 0;
	update_mem = bemps_extra_task_mem(tid);
	tmp_mem = update_mem;
	if(full != 1){
		while(1){
			bool chk_former_task = 0;
			update_mem = bemps_extra_task_mem(tid);
			if(orig_alloc_mem != update_mem){
				chk_former_task = 1;
			}
			if(hipEventQuery(event) == hipSuccess){
				printf("Kernel End\n");
				break;
			}
			if((chk_former_task == 1) && (full != 1)){
				if(update_mem == membytes){
					full = 1;
				}
				if(full == 1){
					priority = -5;
					hipStream_t s_e;
					CUDA_CHECK(hipStreamCreateWithPriority(&s_e, 0, priority));
					printf("Hello\n");
					for(Parameters ret : mem_list){
						CUDA_CHECK(hipMemAdvise(ret.devPtr, ret.alloc_size, ret.advice, ret.device));
						CUDA_CHECK(hipMemPrefetchAsync(ret.devPtr, ret.alloc_size, ret.device, s_e));
					}
					CUDA_CHECK(hipStreamSynchronize(s_e));
					CUDA_CHECK(hipStreamDestroy(s_e));
					break;
				}
			}
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
	int tid = atoi(argv[3]);

	struct timespec specific_time;
	struct tm *now;
	int millsec;
	clock_gettime( CLOCK_REALTIME, &specific_time);
	now = localtime(&specific_time.tv_sec);
	millsec = specific_time.tv_nsec;
  
	millsec = floor (specific_time.tv_nsec/1.0e6);
  
	printf("TID: %d Application begin, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
		now->tm_mon + 1, now->tm_mday, now->tm_hour, 
		now->tm_min, now->tm_sec, millsec);

  printf("WG size of kernel = %d \n", BLOCK_SIZE);

    runTest( argc, argv);

	clock_gettime( CLOCK_REALTIME, &specific_time);
	now = localtime(&specific_time.tv_sec);
	millsec = specific_time.tv_nsec;
  
	millsec = floor (specific_time.tv_nsec/1.0e6);
  
	printf("TID: %d Application end, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
		now->tm_mon + 1, now->tm_mday, now->tm_hour, 
		now->tm_min, now->tm_sec, millsec);

    return EXIT_SUCCESS;
}

void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <max_rows/max_cols> <penalty> \n", argv[0]);
	fprintf(stderr, "\t<dimension>  - x and y dimensions\n");
	fprintf(stderr, "\t<penalty> - penalty(positive integer)\n");
	exit(1);
}

void runTest( int argc, char** argv) 
{
    int max_rows, max_cols, penalty;
    int *input_itemsets, *output_itemsets, *referrence;
	int *matrix_cuda,  *referrence_cuda;
	int size;

	int *tmp_referrence, *tmp_input_itemsets;
	
	int ef_cnt = 0;
  	int ret_dev_id;
	int tid;
	size_t ef_mem;

	Parameters ret1;
	Parameters ret2;
	
    // the lengths of the two sequences should be able to divided by 16.
	// And at current stage  max_rows needs to equal max_cols
	if (argc == 4)
	{
		max_rows = atoi(argv[1]);
		max_cols = atoi(argv[1]);
		penalty = atoi(argv[2]);
		tid = atoi(argv[3]);
	}
    else{
	usage(argc, argv);
    }
	
	if(atoi(argv[1])%16!=0){
	fprintf(stderr,"The dimension values must be a multiple of 16\n");
	exit(1);
	}

	max_rows = max_rows + 1;
	max_cols = max_cols + 1;
	size = max_cols * max_rows;

	dim3 dimGrid;
	dim3 dimBlock(BLOCK_SIZE, 1);
	int block_width = ( max_cols - 1 )/BLOCK_SIZE;
	size = max_cols * max_rows;
	printf("block_width: %d\n", block_width);

	// For reducing host initilization time in task

	tmp_referrence = (int *)malloc( max_rows * max_cols * sizeof(int) );
    tmp_input_itemsets = (int *)malloc( max_rows * max_cols * sizeof(int) );

	// if (!input_itemsets)
	// 	fprintf(stderr, "error: can not allocate memory");

    srand ( 7 );

    // for (int i = 0 ; i < max_cols; i++){
	// 	for (int j = 0 ; j < max_rows; j++){
	// 		tmp_input_itemsets[i*max_cols+j] = 0;
	// 	}
	// }
	
	// printf("Start Needleman-Wunsch\n");
	
	// for( int i=1; i< max_rows ; i++){    //please define your own sequence. 
	// 	tmp_input_itemsets[i*max_cols] = rand() % 10 + 1;
	// }
    // for( int j=1; j< max_cols ; j++){    //please define your own sequence.
	// 	tmp_input_itemsets[j] = rand() % 10 + 1;
	// }


	// for (int i = 1 ; i < max_cols; i++){
	// 	for (int j = 1 ; j < max_rows; j++){
	// 		tmp_referrence[i*max_cols+j] = blosum62[tmp_input_itemsets[i*max_cols]][tmp_input_itemsets[j]];
	// 	}
	// }

    // for( int i = 1; i< max_rows ; i++)
	// 	tmp_input_itemsets[i*max_cols] = -i * penalty;
	// for( int j = 1; j< max_cols ; j++)
    // 	tmp_input_itemsets[j] = -j * penalty;


	// printf("grid_x: %d, grid_y: %d, thread_x: %d, thread_y: %d\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);
	//
	size_t membytes = 0;
	membytes += page_align(sizeof(int)*size);
	membytes += page_align(sizeof(int)*size);
	membytes += 309 * 1024 * 1024;

	struct timespec specific_time;
    struct tm *now;
    int millsec;
    clock_gettime( CLOCK_REALTIME, &specific_time);
    now = localtime(&specific_time.tv_sec);
    millsec = specific_time.tv_nsec;

    millsec = floor (specific_time.tv_nsec/1.0e6);

    printf("TID: %d before schedule, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
        now->tm_mon + 1, now->tm_mday, now->tm_hour, 
        now->tm_min, now->tm_sec, millsec);
	
	long orig_alloc_mem = bemps_begin(tid, block_width, 1, 1, dimBlock.x, dimBlock.y, dimBlock.z, membytes, ret_dev_id);
	
	clock_gettime( CLOCK_REALTIME, &specific_time);
	now = localtime(&specific_time.tv_sec);
	millsec = specific_time.tv_nsec;
	
	millsec = floor (specific_time.tv_nsec/1.0e6);
	
	printf("TID: %d after schedule, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
		now->tm_mon + 1, now->tm_mday, now->tm_hour, 
		now->tm_min, now->tm_sec, millsec);

	if (membytes <= orig_alloc_mem){
		full = 1;
		priority = -5;
	}
	else{
		full = 0;
		priority = 0;
	}

	printf("Full: %d\n", full);

  	printf("ret_dev_id: %d\n", ret_dev_id);

	auto t_start = high_resolution_clock::now();

	CUDA_CHECK(hipMallocManaged(&referrence_cuda, sizeof(int)*size));
	CUDA_CHECK(hipMallocManaged(&input_itemsets, sizeof(int)*size));

	auto t_stop = high_resolution_clock::now();
	auto duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Device mem allocation time: " << duration.count() << std::endl;

	ret1.devPtr = referrence_cuda;
	ret1.advice = hipMemAdviseSetPreferredLocation;
	ret1.device = ret_dev_id;
	ret1.alloc_size = size * sizeof(float);

  	ret2.devPtr = input_itemsets;
	ret2.advice = hipMemAdviseSetPreferredLocation;
	ret2.device = ret_dev_id;
	ret2.alloc_size = size * sizeof(float);

	mem_list.push_back(ret1);
	mem_list.push_back(ret2);

	int fd = -1;
	ssize_t ret = -1;
	size_t test_offset = 2147479552 / 4;

	if(max_rows == 16385){
		printf("Small\n");
		fd = open("small/reference_small.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, referrence_cuda, max_rows * max_cols * sizeof(int));
		printf("%zd\n", ret);
		close(fd);

		fd = open("small/matrix_small.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		ret = -1;
		ret = read(fd, input_itemsets, max_rows * max_cols * sizeof(int));
		printf("%zd\n", ret);
		close(fd);
	}
	else if(max_rows == 32769){
		int cnt = 0;
		size_t alloc_size = size * sizeof(int);
		int quo;
		size_t rem;

		printf("Large\n");
		fd = open("large/reference_large.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		while(1){
			quo = alloc_size / 2147479552;
			printf("quo: %d\n", quo);
			if(quo > 0){
				ret = read(fd, referrence_cuda+(cnt*test_offset), 2147479552);
				printf("%zd\n", ret);
				cnt++;
				alloc_size -= 2147479552;
				if (ret == -1) { 
					error(0,errno,"cannot access tmp.txt");
				}			
			}
			else{
				rem = alloc_size % 2147479552;
				ret = read(fd, referrence_cuda+(cnt*test_offset), rem);
				printf("%zd\n", ret);
				break;
			}
		}
	
		close(fd);

		fd = open("large/matrix_large.txt", O_RDONLY, 0644);
		printf("%d\n", fd);

		cnt = 0;
		alloc_size = size * sizeof(int);

		while(1){
			quo = alloc_size / 2147479552;
			printf("quo: %d\n", quo);
			if(quo > 0){
				ret = read(fd, input_itemsets+(cnt*test_offset), 2147479552);
				printf("%zd\n", ret);
				cnt++;
				alloc_size -= 2147479552;
				if (ret == -1) { 
					error(0,errno,"cannot access tmp.txt");
				}			
			}
			else{
				rem = alloc_size % 2147479552;
				ret = read(fd, input_itemsets+(cnt*test_offset), rem);
				printf("%zd\n", ret);
				break;
			}
		}

		close(fd);
	}

	t_start = high_resolution_clock::now();

	if(full == 1){
		for(Parameters var : mem_list){
			CUDA_CHECK(hipMemAdvise(var.devPtr, var.alloc_size, var.advice, var.device));
			CUDA_CHECK(hipMemPrefetchAsync(var.devPtr, var.alloc_size, var.device, 0));
		}
	}

	// for(Parameters var : mem_list){
	// 	// CUDA_CHECK(hipMemAdvise(var.devPtr, var.alloc_size, var.advice, var.device));
	// 	CUDA_CHECK(hipMemPrefetchAsync(var.devPtr, var.alloc_size, var.device, 0));
	// }
	
	// For reducing host initilization time in task

	// int* dummy;
	// size_t d_size = 13000000000;
	// CUDA_CHECK(hipMalloc(&dummy, d_size));
	// CUDA_CHECK(hipMemset(dummy, 0, d_size));

	size_t free_mem, total_mem;
    // hipMemGetInfo(&free_mem, &total_mem);
 
    // printf("Free: %zd, Total: %zd\n", free_mem, total_mem);

	// CUDA_CHECK(hipMemcpy(referrence_cuda, tmp_referrence, sizeof(int) * size, hipMemcpyHostToDevice));
	// CUDA_CHECK(hipMemcpy(input_itemsets, tmp_input_itemsets, sizeof(int) * size, hipMemcpyHostToDevice));

	// memcpy(referrence_cuda, tmp_referrence, sizeof(int) * size);
	// memcpy(input_itemsets, tmp_input_itemsets, sizeof(int) * size);

	// hipMemGetInfo(&free_mem, &total_mem);

    // printf("Free: %zd, Total: %zd\n", free_mem, total_mem);

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Host to Device memcpy time: " << duration.count() << std::endl;

	printf("Processing top-left matrix\n");
	//process top-left matrix

	if(full == 0){
		clock_gettime( CLOCK_REALTIME, &specific_time);
		now = localtime(&specific_time.tv_sec);
		millsec = specific_time.tv_nsec;

		millsec = floor (specific_time.tv_nsec/1.0e6);

		printf("TID: %d waiting start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
			now->tm_mon + 1, now->tm_mday, now->tm_hour, 
			now->tm_min, now->tm_sec, millsec);

		el_wait(tid);

		clock_gettime( CLOCK_REALTIME, &specific_time);
		now = localtime(&specific_time.tv_sec);
		millsec = specific_time.tv_nsec;

		millsec = floor (specific_time.tv_nsec/1.0e6);

		printf("TID: %d waiting end, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
			now->tm_mon + 1, now->tm_mday, now->tm_hour, 
			now->tm_min, now->tm_sec, millsec);
	}

	t_start = high_resolution_clock::now();

	// if(full == 1){
	// 	clock_gettime( CLOCK_REALTIME, &specific_time);
	// 			now = localtime(&specific_time.tv_sec);
	// 			millsec = specific_time.tv_nsec;

	// 			millsec = floor (specific_time.tv_nsec/1.0e6);

	// 			printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
	// 				now->tm_mon + 1, now->tm_mday, now->tm_hour, 
	// 				now->tm_min, now->tm_sec, millsec);

	// 			nl_signal(tid);

	// 			clock_gettime( CLOCK_REALTIME, &specific_time);
	// 			now = localtime(&specific_time.tv_sec);
	// 			millsec = specific_time.tv_nsec;

	// 			millsec = floor (specific_time.tv_nsec/1.0e6);

	// 			printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
	// 				now->tm_mon + 1, now->tm_mday, now->tm_hour, 
	// 				now->tm_min, now->tm_sec, millsec);
	// }

	for( int i = 1 ; i <= block_width ; i++){
		hipStream_t s1;
		CUDA_CHECK(hipStreamCreateWithPriority(&s1, 0, priority));

		hipEvent_t event1;
		CUDA_CHECK(hipEventCreateWithFlags(&event1, hipEventDisableTiming));

		dimGrid.x = i;
		dimGrid.y = 1;
		needle_cuda_shared_1<<<dimGrid, dimBlock, 0, s1>>>(referrence_cuda, input_itemsets
		                                      ,max_cols, penalty, i, block_width);
		
		CUDA_CHECK(hipEventRecord(event1, s1));

		task_monitoring(event1, tid, orig_alloc_mem, membytes);

		CUDA_CHECK(hipStreamSynchronize(s1));
									
		CUDA_CHECK(hipStreamDestroy(s1));
		CUDA_CHECK(hipEventDestroy(event1));

		if(i == 1){
			if(full == 1){
				clock_gettime( CLOCK_REALTIME, &specific_time);
				now = localtime(&specific_time.tv_sec);
				millsec = specific_time.tv_nsec;

				millsec = floor (specific_time.tv_nsec/1.0e6);

				printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
					now->tm_mon + 1, now->tm_mday, now->tm_hour, 
					now->tm_min, now->tm_sec, millsec);

				nl_signal(tid);

				clock_gettime( CLOCK_REALTIME, &specific_time);
				now = localtime(&specific_time.tv_sec);
				millsec = specific_time.tv_nsec;

				millsec = floor (specific_time.tv_nsec/1.0e6);

				printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
					now->tm_mon + 1, now->tm_mday, now->tm_hour, 
					now->tm_min, now->tm_sec, millsec);
			}
		}
		
		// CUDA_CHECK(hipDeviceSynchronize());
	}

	printf("Processing bottom-right matrix\n");

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "First Kernel execution time: " << duration.count() << std::endl;
	
	t_start = high_resolution_clock::now();

    //process bottom-right matrix
	for( int i = block_width - 1  ; i >= 1 ; i--){
		hipStream_t s2;
		CUDA_CHECK(hipStreamCreateWithPriority(&s2, 0, priority));

		hipEvent_t event2;
		CUDA_CHECK(hipEventCreateWithFlags(&event2, hipEventDisableTiming));

		dimGrid.x = i;
		dimGrid.y = 1;
		needle_cuda_shared_2<<<dimGrid, dimBlock, 0, s2>>>(referrence_cuda, input_itemsets
		                                      ,max_cols, penalty, i, block_width);

		CUDA_CHECK(hipEventRecord(event2, s2));

		task_monitoring(event2, tid, orig_alloc_mem, membytes);

		CUDA_CHECK(hipStreamSynchronize(s2));

		CUDA_CHECK(hipStreamDestroy(s2));
		CUDA_CHECK(hipEventDestroy(event2));

		// if(i == 1){
		// 	if(full == 1){
		// 		clock_gettime( CLOCK_REALTIME, &specific_time);
		// 		now = localtime(&specific_time.tv_sec);
		// 		millsec = specific_time.tv_nsec;

		// 		millsec = floor (specific_time.tv_nsec/1.0e6);

		// 		printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
		// 			now->tm_mon + 1, now->tm_mday, now->tm_hour, 
		// 			now->tm_min, now->tm_sec, millsec);

		// 		nl_signal(tid);

		// 		clock_gettime( CLOCK_REALTIME, &specific_time);
		// 		now = localtime(&specific_time.tv_sec);
		// 		millsec = specific_time.tv_nsec;

		// 		millsec = floor (specific_time.tv_nsec/1.0e6);

		// 		printf("TID: %d sending signal start, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
		// 			now->tm_mon + 1, now->tm_mday, now->tm_hour, 
		// 			now->tm_min, now->tm_sec, millsec);
		// 	}
		// }
		
		// CUDA_CHECK(hipDeviceSynchronize());
	}

	// if(full == 1){
	// 	nl_signal(tid);
	// }
	
	// CUDA_CHECK(hipDeviceSynchronize());

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Second Kernel execution time: " << duration.count() << std::endl;
    // hipMemcpy(output_itemsets, matrix_cuda, sizeof(int) * size, hipMemcpyDeviceToHost);

// 	CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

//   printf("Free: %zd, Total: %zd\n", free_mem, total_mem);
  
  CUDA_CHECK(hipFree(referrence_cuda));

//   CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

//   printf("Free: %zd, Total: %zd\n", free_mem, total_mem);

  if(full == 1){
    ef_mem = page_align(sizeof(int) * size);
    pre_bemps_free(tid, ef_mem);
    ef_cnt = 1;
  }

	t_start = high_resolution_clock::now();

	// CUDA_CHECK(hipMemPrefetchAsync(ret1.devPtr, ret1.alloc_size, hipCpuDeviceId, 0))
	// CUDA_CHECK(hipMemPrefetchAsync(ret2.devPtr, ret2.alloc_size, hipCpuDeviceId, 0))
	// CUDA_CHECK(hipDeviceSynchronize());

	// int *matrix_output;
	// matrix_output = (int*)malloc(max_rows * max_cols * sizeof(int));

	// CUDA_CHECK(hipMemcpy(matrix_output, ret2.devPtr, ret2.alloc_size, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemPrefetchAsync(ret2.devPtr, ret2.alloc_size, hipCpuDeviceId, 0))

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Device to Host memcpy time: " << duration.count() << std::endl;

	t_start = high_resolution_clock::now();

	char* str1 = argv[3];
	char* str2 = "_nongds_output.txt";
	strcat(str1, str2);

	int o_fd = -1;
	o_fd = open(str1, O_CREAT | O_RDWR, 0664);

	if (o_fd < 0) {
		std::cerr << "file open error:" << std::endl;
	}

	printf("%d\n", o_fd);

	int cnt = 0;
	size_t alloc_size = size * sizeof(int);
	int quo;
	size_t rem;
	ssize_t o_ret = -1;

	while(1){
		quo = alloc_size / 2147479552;
		printf("quo: %d\n", quo);
		if(quo > 0){
			o_ret = write(o_fd, input_itemsets+(cnt*test_offset), 2147479552);
			printf("%zd\n", o_ret);
			cnt++;
			alloc_size -= 2147479552;
			if (o_ret == -1) { 
				error(0,errno,"cannot access tmp.txt");
			}			
		}
		else{
			rem = alloc_size % 2147479552;
			o_ret = write(o_fd, input_itemsets+(cnt*test_offset), rem);
			printf("%zd\n", o_ret);
			break;
		}
	}

	close(o_fd);

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Host to Storage memcpy time: " << duration.count() << std::endl;

	t_start = high_resolution_clock::now();

	hipFree(input_itemsets);

	t_stop = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(t_stop - t_start);
	std::cout << "Tid: " << tid << " Device memory deallocation time: " << duration.count() << std::endl;

	bemps_free(tid);

	clock_gettime( CLOCK_REALTIME, &specific_time);
	now = localtime(&specific_time.tv_sec);
	millsec = specific_time.tv_nsec;

	millsec = floor (specific_time.tv_nsec/1.0e6);

	printf("TID: %d finish work, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
        now->tm_mon + 1, now->tm_mday, now->tm_hour, 
        now->tm_min, now->tm_sec, millsec);

	// if (o_fd < 0) {
	// 	std::cerr << "file open error:" << std::endl;
	// }

	// printf("%d\n", o_fd);

	// ssize_t o_ret = -1;
	// o_ret = write(o_fd, matrix_output, max_rows * max_cols * sizeof(int));
	
	// printf("%zd\n", o_ret);

	// close(o_fd);

//#define TRACEBACK
#ifdef TRACEBACK
	
FILE *fpo = fopen("result.txt","w");
fprintf(fpo, "print traceback value GPU:\n");

// CUDA_CHECK(hipMemPrefetchAsync(ret1.devPtr, ret1.alloc_size, hipCpuDeviceId, 0))
CUDA_CHECK(hipMemPrefetchAsync(ret2.devPtr, ret2.alloc_size, hipCpuDeviceId, 0))
// CUDA_CHECK(hipDeviceSynchronize());

for (int i = max_rows - 2,  j = max_rows - 2; i>=0, j>=0;){
	int nw, n, w, traceback;
	if ( i == max_rows - 2 && j == max_rows - 2 )
		fprintf(fpo, "%d ", input_itemsets[ i * max_cols + j]); //print the first element
	if ( i == 0 && j == 0 )
	   break;
	if ( i > 0 && j > 0 ){
		nw = input_itemsets[(i - 1) * max_cols + j - 1];
		w  = input_itemsets[ i * max_cols + j - 1 ];
		n  = input_itemsets[(i - 1) * max_cols + j];
	}
	else if ( i == 0 ){
		nw = n = LIMIT;
		w  = input_itemsets[ i * max_cols + j - 1 ];
	}
	else if ( j == 0 ){
		nw = w = LIMIT;
		n  = input_itemsets[(i - 1) * max_cols + j];
	}
	else{
	}

	//traceback = maximum(nw, w, n);
	int new_nw, new_w, new_n;
	new_nw = nw + referrence_cuda[i * max_cols + j];
	new_w = w - penalty;
	new_n = n - penalty;
	
	traceback = maximum(new_nw, new_w, new_n);
	if(traceback == new_nw)
		traceback = nw;
	if(traceback == new_w)
		traceback = w;
	if(traceback == new_n)
		traceback = n;
		
	fprintf(fpo, "%d ", traceback);

	if(traceback == nw )
	{i--; j--; continue;}

	else if(traceback == w )
	{j--; continue;}

	else if(traceback == n )
	{i--; continue;}

	else
	;
}

fclose(fpo);

#endif
	// t_start = high_resolution_clock::now();

	// hipFree(referrence_cuda);
	// hipFree(input_itemsets);

	// t_stop = high_resolution_clock::now();
	// duration = duration_cast<milliseconds>(t_stop - t_start);
	// std::cout << "Tid: " << tid << " Device memory deallocation time: " << duration.count() << std::endl;

	// bemps_free(tid);

	// clock_gettime( CLOCK_REALTIME, &specific_time);
	// now = localtime(&specific_time.tv_sec);
	// millsec = specific_time.tv_nsec;

	// millsec = floor (specific_time.tv_nsec/1.0e6);

	// printf("TID: %d finish work, [%04d/%02d/%02d] %02d:%02d:%02d msec : %d\n", tid, 1900 + now->tm_year, 
    //     now->tm_mon + 1, now->tm_mday, now->tm_hour, 
    //     now->tm_min, now->tm_sec, millsec);

	// free(referrence);
	// free(input_itemsets);
	// free(output_itemsets);
	
}

